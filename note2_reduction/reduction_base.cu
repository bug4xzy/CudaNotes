#include <stdio.h>
#include <hip/hip_runtime.h>

#include <time.h>
#include <limits>

// CUDA错误检查宏
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA Error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
            exit(1); \
        } \
    } while (0)

// GPU Kernel
template<typename T, int blockSize>
__global__ void reduce_base(T *d_in, T *d_out, int N) {
    __shared__ T sdata[blockSize];
    int tid = threadIdx.x;
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // 使用Kahan求和算法的局部累加
    T sum = T(0);
    T c = T(0);
    
    // 每个线程处理多个元素
    for(int i = globalIdx; i < N; i += blockDim.x * gridDim.x) {
        T y = d_in[i] - c;
        T t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }
    
    // 加载局部和到共享内存
    sdata[tid] = sum;
    __syncthreads();
    
    // 在共享内存中进行规约
    for(int s = blockDim.x/2; s > 0; s >>= 1) {
        if(tid < s) {
            // 对共享内存中的数据进行Kahan求和
            T y = sdata[tid + s] - c;
            T t = sdata[tid] + y;
            c = (t - sdata[tid]) - y;
            sdata[tid] = t;
        }
        __syncthreads();
    }
    
    // 将每个block的结果写入全局内存
    if(tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}

// CPU规约函数
template<typename T>
void reduce_sum_cpu_kahan(T *x, T *out, int N) {
    T sum = T(0);
    T c = T(0);
    
    for (int i = 0; i < N; i++) {
        T y = x[i] - c;
        T t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }
    
    out[0] = sum;
}

// 最终规约函数
template<typename T>
T final_reduction_kahan(T *partial_sums, int size) {
    T sum = T(0);
    T c = T(0);
    
    for (int i = 0; i < size; i++) {
        T y = partial_sums[i] - c;
        T t = sum + y;
        c = (t - sum) - y;
        sum = t;
    }
    
    return sum;
}

// 数据初始化函数
template<typename T>
void initialize_data(T *x, int N) {
    srand(time(NULL));
    for (int i = 0; i < N; i++) {
        x[i] = static_cast<T>(rand() % 1000) / static_cast<T>(100);
    }
}

// 结果验证函数
template<typename T>
bool verify_results(T *gpu_result, T *cpu_result, int N) {
    T relative_error = std::abs(cpu_result[0] - gpu_result[0]) / cpu_result[0];
    if (relative_error > std::numeric_limits<T>::epsilon() * 100) {
        printf("Result verification failed!\n");
        printf("Relative Error: %e\n", static_cast<double>(relative_error));
        return false;
    }
    return true;
}

// 资源清理函数
template<typename T>
void cleanup(T *d_in, T *d_out, T *h_in, T *h_out, T *h_cpu_out) {
    if (d_in) hipFree(d_in);
    if (d_out) hipFree(d_out);
    if (h_in) free(h_in);
    if (h_out) free(h_out);
    if (h_cpu_out) free(h_cpu_out);
}

// 主计算函数
template<typename T>
void run_reduction(int N) {
    const int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    size_t nbytes = N * sizeof(T);
    size_t out_nbytes = gridSize * sizeof(T);

    // 声明指针
    T *d_in = NULL, *d_out = NULL;
    T *h_in = NULL, *h_out = NULL, *h_cpu_out = NULL;

    // 分配内存
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_in, nbytes));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_out, out_nbytes));

    h_in = (T *)malloc(nbytes);
    h_out = (T *)malloc(out_nbytes);
    h_cpu_out = (T *)malloc(sizeof(T));
    
    if (!h_in || !h_out || !h_cpu_out) {
        printf("CPU Memory allocation failed!\n");
        cleanup(d_in, d_out, h_in, h_out, h_cpu_out);
        return;
    }

    // 初始化数据
    initialize_data(h_in, N);

    // 复制数据到GPU并初始化输出数组
    CHECK_CUDA_ERROR(hipMemcpy(d_in, h_in, nbytes, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemset(d_out, 0, out_nbytes));

    // GPU计时
    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    
    float gpu_time = 0.0f;
    CHECK_CUDA_ERROR(hipEventRecord(start));
    
    // 调用kernel
    reduce_base<T, blockSize><<<gridSize, blockSize>>>(d_in, d_out, N);
    
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&gpu_time, start, stop));
    
    // 检查kernel执行
    CHECK_CUDA_ERROR(hipGetLastError());
    
    // 复制结果回CPU
    CHECK_CUDA_ERROR(hipMemcpy(h_out, d_out, out_nbytes, hipMemcpyDeviceToHost));

    // CPU计算参考结果
    reduce_sum_cpu_kahan(h_in, h_cpu_out, N);

    // 使用Kahan求和算法对block结果进行最终规约
    T final_sum = T(0);
    T c = T(0);
    for(int i = 0; i < gridSize; i++) {
        T y = h_out[i] - c;
        T t = final_sum + y;
        c = (t - final_sum) - y;
        final_sum = t;
    }
    h_out[0] = final_sum;

    // 验证结果
    if (verify_results(h_out, h_cpu_out, 1)) {
        printf("Results verified successfully!\n");
        printf("GPU Execution time: %f ms\n", gpu_time);
        printf("Final sum: %e\n", static_cast<double>(h_out[0]));
    }
    else {
        printf("Results verification failed!\n");
        printf("CPU result: %e\n", static_cast<double>(h_cpu_out[0]));
        printf("GPU result: %e\n", static_cast<double>(h_out[0]));
    }

    // 清理资源
    cleanup(d_in, d_out, h_in, h_out, h_cpu_out);
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));
}

int main() {
    const int N = 25600000;
    
    printf("\nRunning with float:\n");
    run_reduction<float>(N);
    
    printf("\nRunning with double:\n");
    run_reduction<double>(N);
    
    printf("\nRunning with int:\n");
    run_reduction<int>(N);
    
    return 0;
}